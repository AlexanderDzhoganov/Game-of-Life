#include "hip/hip_runtime.h"
#include "../include/headers.h"

typedef unsigned char uint8_t;

texture<uint8_t, 2, hipReadModeElementType> inputTex;

__global__ void gameOfLife(gof_data* gof) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	uint8_t p = tex2D(inputTex, x, y);
	uint8_t neighbors = 0;
	neighbors += tex2D(inputTex, x+1, y);
	neighbors += tex2D(inputTex, x-1, y);
	neighbors += tex2D(inputTex, x, y+1);
	neighbors += tex2D(inputTex, x, y-1);
	neighbors += tex2D(inputTex, x+1, y+1);
	neighbors += tex2D(inputTex, x-1, y-1);
	neighbors += tex2D(inputTex, x-1, y+1);
	neighbors += tex2D(inputTex, x+1, y-1);

	__syncthreads();

	uint8_t final = 0;
			
	if(p != 0) {
		for(int i=0; i<gof->S_n; i++) {
			if(neighbors == gof->S[i]) {
				final = 1;
			}
		}
	} else {
		for(int i=0; i<gof->B_n; i++) {
			if(neighbors == gof->B[i]) {
				final = 1;
				break;
			}
		}
	}

	__syncthreads();
	gof->outputBuffer[x+y*gof->width] = final;
}

CUDA::CUDA(int width_, int height_) {
	width = width_;
	height = height_;
	bufferSize = sizeof(uint8_t)*width*height;
	hipMalloc((void**)&cudaOutputBuffer, bufferSize);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipMallocArray(&cudaInputArray, &channelDesc, width, height);
	inputTex.addressMode[0] = hipAddressModeWrap;
    inputTex.addressMode[1] = hipAddressModeWrap;
    inputTex.filterMode = hipFilterModePoint;
    inputTex.normalized = false;
	hipBindTextureToArray(inputTex, cudaInputArray, channelDesc);
}

CUDA::~CUDA() {

}

void CUDA::getInput(uint8_t* buffer) {
	cudaMemcpyToArrayAsync(cudaInputArray, 0, 0, buffer, bufferSize, hipMemcpyHostToDevice); 
}

void CUDA::runGameOfLife(uint8_t* returnBuffer, int* B, int B_n, int* S, int S_n) {
	dim3 grid(width/16, height/16);
	dim3 block(16, 16);

	gof_data* gof;
	hipMalloc((void**)&gof, sizeof(gof_data));

	gof_data l_gof;
	l_gof.inputBuffer = cudaInputBuffer;
	l_gof.outputBuffer = cudaOutputBuffer;
	l_gof.height = height;
	l_gof.width = width;

	memcpy(l_gof.B, B, sizeof(int)*B_n);
	memcpy(l_gof.S, S, sizeof(int)*S_n);
	l_gof.B_n = B_n;
	l_gof.S_n = S_n;

	hipMemcpy(gof, &l_gof, sizeof(gof_data), hipMemcpyHostToDevice);
	gameOfLife <<< grid, block >>> (gof);
	hipMemcpy(returnBuffer, cudaOutputBuffer, bufferSize, hipMemcpyDeviceToHost);
}